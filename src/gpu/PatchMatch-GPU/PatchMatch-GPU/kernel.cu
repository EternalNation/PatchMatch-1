#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <iostream>
#include <cmath>

using namespace cv;
using namespace std;

const int patch_w = 5;
int	pm_iters = 5;
int rs_max = INT_MAX;

dim3 threadsPerBlock(patch_w,patch_w);

#define XY_TO_INT(x, y) (((y)<<12)|(x))
#define INT_TO_X(v) ((v)&((1<<12)-1))
#define INT_TO_Y(v) ((v>>12)&((1<<12)-1))

//l2 distance between two patches
__global__ void dist_gpu(hipPitchedPtr a, hipPitchedPtr b, int * params){//params : 0 - ax, 1 - ay, 2 - bx, 3 - by, 4 - res,
	//__shared__ int res[patch_w*patch_w];
	int dr = a.ptr[params[1] + threadIdx.y][params[0] + threadIdx.x][2] - b[by + threadIdx.y][bx + threadIdx.x][2];
	int dg = a.ptr[params[1] + threadIdx.y][params[0] + threadIdx.x][1] - b[by + threadIdx.y][bx + threadIdx.x][1];
	int db = a.ptr[params[1] + threadIdx.y][params[0] + threadIdx.x][0] - b[by + threadIdx.y][bx + threadIdx.x][0];
	/*patchd[threadIdx.y][threadIdx.x] = dr*dr + dg*dg + db*db;*/
	res[threadIdx.y*patch_w+threadIdx.x] = dr*dr + dg*dg + db*db;
	//__syncthreads();
	//int i = patch_w*patch_w / 2;
	//int j = patch_w*patch_w % 2;
	//while (i != 0)
	//{
	//	if (threadIdx.y*patch_w + threadIdx.x<i){
	//		res[threadIdx.y*patch_w + threadIdx.x] += res[threadIdx.y*patch_w + threadIdx.x + i];
	//	}
	//	if (j == 1 && threadIdx.y*patch_w + threadIdx.x == i - 1){
	//		res[threadIdx.y*patch_w + threadIdx.x] += res[threadIdx.y*patch_w + threadIdx.x + i + 1];
	//	}
	//	__syncthreads();
	//	j = i % 2;
	//	i = i / 2;
	//}
	//if (threadIdx.x == 0 && threadIdx.y == 0){
	//	//cout << "Total result is " << res[threadIdx.y*patch_w + threadIdx.x] << endl;
	//	if (res[threadIdx.y*patch_w + threadIdx.x] >= cutoff){
	//		ans = cutoff;
	//	}
	//	else{
	//		ans = res[threadIdx.y*patch_w + threadIdx.x];
	//	}
	//}
	/*ans = 0;
	for (int dy = 0; dy < patch_w; dy++) {
		for (int dx = 0; dx < patch_w; dx++) {
			int dr = a[ay + dy][ax + dx][2] - b[by + dy][bx + dx][2];
			int dg = a[ay + dy][ax + dx][1] - b[by + dy][bx + dx][1];
			int db = a[ay + dy][ax + dx][0] - b[by + dy][bx + dx][0];
			ans += dr*dr + dg*dg + db*db;
		}
		if (ans >= cutoff) { return cutoff; }
	}
	return ans;*/
	
}

int dist_test(int ***a, int ***b, int ax, int ay, int bx, int by, int cutoff = INT_MAX){
	int ans = 0;
	for (int dy = 0; dy < patch_w; dy++) {
		for (int dx = 0; dx < patch_w; dx++) {
			int dr = a[ay + dy][ax + dx][2] - b[by + dy][bx + dx][2];
			int dg = a[ay + dy][ax + dx][1] - b[by + dy][bx + dx][1];
			int db = a[ay + dy][ax + dx][0] - b[by + dy][bx + dx][0];
			ans += dr*dr + dg*dg + db*db;
		}
		if (ans >= cutoff) { return cutoff; }
	}
	return ans;
}

//int dist(Mat a, Mat b, int ax, int ay, int bx, int by, int cutoff = INT_MAX){
//	int ans = 0;
//	for (int dy = 0; dy < patch_w; dy++) {
//		for (int dx = 0; dx < patch_w; dx++) {
//			Vec3b ai = a.at<Vec3b>(ay + dy, ax + dx);
//			Vec3b bi = b.at<Vec3b>(by + dy, bx + dx);
//			int dr = abs(ai.val[2] - bi.val[2]);
//			int dg = abs(ai.val[1] - bi.val[1]);
//			int db = abs(ai.val[0] - bi.val[0]);
//			ans += dr*dr + dg*dg + db*db;
//		}
//		if (ans >= cutoff) { return cutoff; }
//	}
//	return ans;
//}

void improve_guess(int *** a, int *** b, int ax, int ay, int &xbest, int &ybest, int &dbest, int bx, int by) {
	int d_gpu[patch_w*patch_w] = { 0 }, d_cpu = 0;
	dist_gpu<<<1, threadsPerBlock >>>(a, b, ax, ay, bx, by, d_gpu, dbest);
	d_cpu = dist_test(a, b, ax, ay, bx, by, dbest);
	int resgpu = 0;
	for (int i = 0; i < patch_w; i++)
	{
		resgpu += d_gpu[i];
	}
	cout << "d-gpu is :" << resgpu << ", d-cpu:" << d_cpu << endl;
	//int d = dist_test(a, b, ax, ay, bx, by, dbest);
	if (d_cpu < dbest) {
		dbest = d_cpu;
		xbest = bx;
		ybest = by;
	}
}

//get the approximate nearest neighbor and set it into ann
void patchmatch(Mat a, Mat b, unsigned int **&ann, int **&annd) {
	
	/* Initialize with random nearest neighbor field (NNF). */
	int ans = 0;
	int aew = a.cols - patch_w + 1, aeh = b.rows - patch_w + 1;       /* Effective width and height (possible upper left corners of patches). */
	int bew = b.cols - patch_w + 1, beh = b.rows - patch_w + 1;

	int *** a_pixel = new int **[a.rows];//set the rgb value from matrix a in a_pixel
	int *** b_pixel = new int **[b.rows];
	int *** dev_a;// device variable of a_pixel
	int *** dev_b;

	int params[5] = { 0 }; // 0 - ax, 1 - ay, 2 - bx, 3 - by, 4 - res
	int * dev_params;

	ann = new unsigned int *[a.rows];
	annd = new int *[a.rows];

	// initialize ann, annd ,a_pixel, b_pixel
	for (int i = 0; i < a.rows; i++)
	{
		ann[i] = new unsigned int[a.cols];
		annd[i] = new int[a.cols];
		a_pixel[i] = new int*[a.cols];
		memset(ann[i], 0, a.cols);
		memset(annd[i], 0, a.cols);
		for (int j = 0; j < a.cols; j++)
		{
			Vec3b ai = a.at<Vec3b>(i, j);
			a_pixel[i][j] = new int[3];
			a_pixel[i][j][0] = (int)ai.val[0];
			a_pixel[i][j][1] = (int)ai.val[1];
			a_pixel[i][j][2] = (int)ai.val[2];
		}
	}
	for (int i = 0; i < b.rows; i++)
	{
		b_pixel[i] = new int*[b.cols];
		for (int j = 0; j < b.cols; j++)
		{
			Vec3b bi = b.at<Vec3b>(i, j);
			b_pixel[i][j] = new int[3];
			b_pixel[i][j][0] = (int)bi.val[0];
			b_pixel[i][j][1] = (int)bi.val[1];
			b_pixel[i][j][2] = (int)bi.val[2];
		}
	}

	// cuda malloc
	hipError_t cudaStatus;
	//3 dims
	hipExtent a_extent = make_hipExtent(a.cols, a.rows, 3);
	hipExtent b_extent = make_hipExtent(b.cols, b.rows, 3);
	hipPitchedPtr a_devPitchedPtr;
	hipPitchedPtr b_devPitchedPtr;

	hipMalloc3D(&a_devPitchedPtr, a_extent);
	hipMemcpy3DParms a_HostToDev = { 0 };
	a_HostToDev.srcPtr = make_hipPitchedPtr((void*)a_pixel, a.cols * sizeof(int), a.cols, a.rows);
	a_HostToDev.dstPtr = a_devPitchedPtr;
	a_HostToDev.extent = a_extent;
	a_HostToDev.kind = hipMemcpyHostToDevice;
	cudaStatus = hipMemcpy3D(&a_HostToDev);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "MemcpyHtD: %s\n", hipGetErrorString(cudaStatus));
	}

	hipMalloc3D(&b_devPitchedPtr, b_extent);
	hipMemcpy3DParms b_HostToDev = { 0 };
	b_HostToDev.srcPtr = make_hipPitchedPtr((void*)b_pixel, b.cols * sizeof(int), b.cols, b.rows);
	b_HostToDev.dstPtr = b_devPitchedPtr;
	b_HostToDev.extent = b_extent;
	b_HostToDev.kind = hipMemcpyHostToDevice;
	cudaStatus = hipMemcpy3D(&b_HostToDev);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "MemcpyHtD: %s\n", hipGetErrorString(cudaStatus));
	}

	

	for (int ay = 0; ay < aeh; ay++) {
		for (int ax = 0; ax < aew; ax++) {
			int bx = rand() % bew;
			int by = rand() % beh;
			
			params[0] = ax;
			params[1] = ay;
			params[2] = bx;
			params[3] = by;

			hipMalloc((void**)&dev_params, 5 * sizeof(int));
			hipMemcpy(dev_params, params, 5 * sizeof(int), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess){
				fprintf(stderr, "MemcpyHtD: %s\n", hipGetErrorString(cudaStatus));
			}
			dist_gpu<<<1, threadsPerBlock>>>(dev_a, dev_b, dev_params);

			cudaStatus = hipMemcpy(params, dev_params, 5 * sizeof(int), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess){
				fprintf(stderr, "MemcpyDtH: %s\n", hipGetErrorString(cudaStatus));
			}
			ann[ay][ax] = XY_TO_INT(bx, by);
			annd[ay][ax] = dev_params[4];

			hipFree(dev_params);

			//annd[ay][ax] = dist_test(a_pixel, b_pixel , ax, ay, bx, by);
			//cout << "ann :" << ann[ay][ax] << ", annd :" << annd[ay][ax]<<endl;
			ans = 0;
			
			
		}
	}
	for (int iter = 0; iter < pm_iters; iter++) {
		/* In each iteration, improve the NNF, by looping in scanline or reverse-scanline order. */
		int ystart = 0, yend = aeh, ychange = 1;
		int xstart = 0, xend = aew, xchange = 1;
		if (iter % 2 == 1) {
			xstart = xend - 1; xend = -1; xchange = -1;
			ystart = yend - 1; yend = -1; ychange = -1;
		}
		for (int ay = ystart; ay != yend; ay += ychange) {
			for (int ax = xstart; ax != xend; ax += xchange) {
				/* Current (best) guess. */
				unsigned int v = ann[ay][ax];
				int xbest = INT_TO_X(v), ybest = INT_TO_Y(v);
				int dbest = annd[ay][ax];

				/* Propagation: Improve current guess by trying instead correspondences from left and above (below and right on odd iterations). */
				if ((unsigned)(ax - xchange) < (unsigned)aew && (ax - xchange) >= 0) {
					int vp = ann[ay][ax - xchange];
					int xp = INT_TO_X(vp) + xchange, yp = INT_TO_Y(vp);
					if ((unsigned)xp < (unsigned)bew) {
						//improve guress
						params[0] = ax;
						params[1] = ay;
						params[2] = xp;
						params[3] = yp;

						hipMalloc((void**)&dev_params, 5 * sizeof(int));
						hipMemcpy(dev_params, params, 5 * sizeof(int), hipMemcpyHostToDevice);
						if (cudaStatus != hipSuccess){
							fprintf(stderr, "MemcpyHtD: %s\n", hipGetErrorString(cudaStatus));
						}
						dist_gpu<<<1, threadsPerBlock>>>(dev_a, dev_b, dev_params);

						cudaStatus = hipMemcpy(params, dev_params, 5 * sizeof(int), hipMemcpyDeviceToHost);
						if (cudaStatus != hipSuccess){
							fprintf(stderr, "MemcpyDtH: %s\n", hipGetErrorString(cudaStatus));
						}
						if (dev_params[4] < dbest){
							xbest = xp;
							ybest = yp;
							dbest = dev_params[4];
						}

						hipFree(dev_params);

						//improve_guess(a_pixel, b_pixel, ax, ay, xbest, ybest, dbest, xp, yp);
					}
				}

				if ((unsigned)(ay - ychange) < (unsigned)aeh && (ay - ychange) >= 0) {
					int vp = ann[ay - ychange][ax];
					int xp = INT_TO_X(vp), yp = INT_TO_Y(vp) + ychange;
					if ((unsigned)yp < (unsigned)beh) {
						//improve guress
						params[0] = ax;
						params[1] = ay;
						params[2] = xp;
						params[3] = yp;

						hipMalloc((void**)&dev_params, 5 * sizeof(int));
						hipMemcpy(dev_params, params, 5 * sizeof(int), hipMemcpyHostToDevice);
						if (cudaStatus != hipSuccess){
							fprintf(stderr, "MemcpyHtD: %s\n", hipGetErrorString(cudaStatus));
						}
						dist_gpu<<<1, threadsPerBlock>>>(dev_a, dev_b, dev_params);

						cudaStatus = hipMemcpy(params, dev_params, 5 * sizeof(int), hipMemcpyDeviceToHost);
						if (cudaStatus != hipSuccess){
							fprintf(stderr, "MemcpyDtH: %s\n", hipGetErrorString(cudaStatus));
						}
						if (dev_params[4] < dbest){
							xbest = xp;
							ybest = yp;
							dbest = dev_params[4];
						}

						hipFree(dev_params);
						//improve_guess(a_pixel, b_pixel, ax, ay, xbest, ybest, dbest, xp, yp);
					}
				}

				/* Random search: Improve current guess by searching in boxes of exponentially decreasing size around the current best guess. */
				int rs_start = rs_max;
				if (rs_start > MAX(b.cols, b.rows)) { rs_start = MAX(b.cols, b.rows); }
				for (int mag = rs_start; mag >= 1; mag /= 2) {
					/* Sampling window */
					int xmin = MAX(xbest - mag, 0), xmax = MIN(xbest + mag + 1, bew);
					int ymin = MAX(ybest - mag, 0), ymax = MIN(ybest + mag + 1, beh);
					int xp = xmin + rand() % (xmax - xmin);
					int yp = ymin + rand() % (ymax - ymin);
					//improve_guess(a_pixel, b_pixel, ax, ay, xbest, ybest, dbest, xp, yp);
					//improve guress
					params[0] = ax;
					params[1] = ay;
					params[2] = xp;
					params[3] = yp;

					hipMalloc((void**)&dev_params, 5 * sizeof(int));
					hipMemcpy(dev_params, params, 5 * sizeof(int), hipMemcpyHostToDevice);
					if (cudaStatus != hipSuccess){
						fprintf(stderr, "MemcpyHtD: %s\n", hipGetErrorString(cudaStatus));
					}
					dist_gpu<<<1, threadsPerBlock>>>(dev_a, dev_b, dev_params);

					cudaStatus = hipMemcpy(params, dev_params, 5 * sizeof(int), hipMemcpyDeviceToHost);
					if (cudaStatus != hipSuccess){
						fprintf(stderr, "MemcpyDtH: %s\n", hipGetErrorString(cudaStatus));
					}
					if (dev_params[4] < dbest){
						xbest = xp;
						ybest = yp;
						dbest = dev_params[4];
					}

					hipFree(dev_params);
				}

				ann[ay][ax] = XY_TO_INT(xbest, ybest);
				annd[ay][ax] = dbest;
			}
		}
	}
}

int dist_p(Mat a, Mat b, int ax, int ay, int bx, int by){
	Vec3b ai = a.at<Vec3b>(ay, ax);
	Vec3b bi = b.at<Vec3b>(by, bx);
	int dr = abs(ai.val[2] - bi.val[2]);
	int dg = abs(ai.val[1] - bi.val[1]);
	int db = abs(ai.val[0] - bi.val[0]);
	return dr*dr + dg*dg + db*db;
}

/* nearest voting */
Mat reconstruct(Mat a, Mat b, unsigned int ** ann){
	Mat a_recon;
	a.copyTo(a_recon);
	int aew = a.cols - patch_w + 1, aeh = b.rows - patch_w + 1;       /* Effective width and height (possible upper left corners of patches). */
	int bew = b.cols - patch_w + 1, beh = b.rows - patch_w + 1;
	int ystart = 0, yend = aeh, ychange = 1;
	int xstart = 0, xend = aew, xchange = 1;
	unsigned int ybest = 0, xbest = 0, v = 0;
	//difference of pixel
	unsigned int ** pnnd;
	unsigned int ** pnn;
	pnn = new unsigned int *[a.rows];
	pnnd = new unsigned int *[a.rows];
	for (int i = 0; i < a.rows; i++)
	{
		pnn[i] = new unsigned int[a.cols];
		pnnd[i] = new unsigned int[a.cols];
		memset(pnn[i], 0, a.cols);
	}

	
	for (int ay = 0; ay < aeh; ay++) {
		for (int ax = 0; ax < aew; ax++) {
			
			if (ay < aeh&&ax < aew)
			{
				pnn[ay][ax] = ann[ay][ax];
				v = ann[ay][ax];
				xbest = INT_TO_X(v);
				ybest = INT_TO_Y(v);
				pnnd[ay][ax] = dist_p(a, b, ax, ay, xbest, ybest);
			}
			else if (ay >= aeh&&ax < aew){
				v = ann[aeh-1][ax];
				xbest = INT_TO_X(v);
				ybest = INT_TO_Y(v);
				ybest += (ay - aeh+1);
				pnn[ay][ax] = XY_TO_INT(xbest, ybest);
				pnnd[ay][ax] = dist_p(a, b, ax, ay, xbest, ybest);
			}
			else if (ay < aeh&&ax >= aew){
				v = ann[ay][aew-1];
				xbest = INT_TO_X(v);
				ybest = INT_TO_Y(v);
				xbest += (ax - aew+1);
				pnn[ay][ax] = XY_TO_INT(xbest, ybest);
				pnnd[ay][ax] = dist_p(a, b, ax, ay, xbest, ybest);
		
			}
			else{
				v = ann[aeh-1][aew-1];
				xbest = INT_TO_X(v);
				ybest = INT_TO_Y(v);
				xbest += (ax - aew+1);
				ybest += (ay - aeh+1);
				pnn[ay][ax] = XY_TO_INT(xbest, ybest);
				pnnd[ay][ax] = dist_p(a, b, ax, ay, xbest, ybest);
			}

		}
	}


	for (int ay = ystart; ay != yend; ay += ychange) {
		for (int ax = xstart; ax != xend; ax += xchange) {
			v = ann[ay][ax];
			xbest = INT_TO_X(v);
			ybest = INT_TO_Y(v);

			for (int dy = 0; dy < patch_w; dy++) {
				for (int dx = 0; dx < patch_w; dx++) {
					if (pnnd[ay + dy][ax + dx]>dist_p(a, b, ax + dx, ay + dy, xbest + dx ,ybest + dy)){
						pnn[ay + dy][ax + dx] = XY_TO_INT(xbest + dx, ybest + dy);
						pnnd[ay + dy][ax + dx] = dist_p(a, b, ax + dx, ay + dy, xbest + dx, ybest + dy);
					}

				}

			}
		}
	}

	for (int ay = ystart; ay < a.rows;ay++){
		for (int ax = xstart; ax < a.cols; ax++)
		{
			v = pnn[ay][ax];
			xbest = INT_TO_X(v);
			ybest = INT_TO_Y(v);

			Vec3b bi = b.at<Vec3b>(ybest, xbest);
			a_recon.at<Vec3b>(ay, ax).val[2] = bi.val[2];
			a_recon.at<Vec3b>(ay, ax).val[1] = bi.val[1];
			a_recon.at<Vec3b>(ay, ax).val[0] = bi.val[0];
		}
	}
	return a_recon;
}

//
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}

int main()
{

	String window_name = "reconstructed";

	// define img matrix
	Mat a = imread("Image/disp1.png");
	Mat b = imread("Image/view1.png");
	Mat a_recon;
	if (a.empty()||b.empty())
	{	
		cout << "image cannot read!" << endl;
		waitKey();
		exit;
	}
	// define and initialize ann and annd array
	int **annd;
	unsigned int**ann;
	patchmatch(a, b, ann, annd);

	a_recon = reconstruct(a, b, ann);

	namedWindow(window_name, CV_WINDOW_AUTOSIZE);
	imshow(window_name, a_recon);

	namedWindow("a", CV_WINDOW_AUTOSIZE);
	imshow("a", a);

	namedWindow("b", CV_WINDOW_AUTOSIZE);
	imshow("b", b);

	//imwrite("Image/result.png", a_recon);

	waitKey();

	destroyAllWindows();

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
